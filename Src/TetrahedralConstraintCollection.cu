#include "hip/hip_runtime.h"

#include "Node.h"
#include "TetrahedralConstraintCollection.h"
#include "hip/hip_runtime.h"
#include ""

#include <svd3x3/svd3_cuda.h>

#define GLM_FORCE_CUDA
#include <glm/glm.hpp>
#include <glm/gtc/matrix_access.hpp>
#include <glm/gtc/matrix_inverse.hpp>
#include <glm/gtc/matrix_transform.hpp>

__global__ void projectTets(
    Pies::TetrahedralConstraint* devTets,
    glm::vec3* devNodePositions,
    glm::mat3x4* wAtBp,
    int tetCount) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= tetCount) {
    return;
  }

  const Pies::TetrahedralConstraint& tet = devTets[i];

  const glm::vec3& x1 = devNodePositions[tet.nodeIds[0]];
  const glm::vec3& x2 = devNodePositions[tet.nodeIds[1]];
  const glm::vec3& x3 = devNodePositions[tet.nodeIds[2]];
  const glm::vec3& x4 = devNodePositions[tet.nodeIds[3]];

  glm::mat3 P(x2 - x1, x3 - x1, x4 - x1);

  // Deformation gradient
  glm::mat3 F = glm::transpose(P * tet.Qinv);

  glm::mat3 U(0.0f);
  glm::vec3 sigma(0.0f);
  glm::mat3 V(0.0f);

  svd(
      // Remember glm is stored column major
      F[0][0],
      F[1][0],
      F[2][0],
      F[0][1],
      F[1][1],
      F[2][1],
      F[0][2],
      F[1][2],
      F[2][2],
      U[0][0],
      U[1][0],
      U[2][0],
      U[0][1],
      U[1][1],
      U[2][1],
      U[0][2],
      U[1][2],
      U[2][2],
      sigma[0],
      sigma[1],
      sigma[2],
      V[0][0],
      V[1][0],
      V[2][0],
      V[0][1],
      V[1][1],
      V[2][1],
      V[0][2],
      V[1][2],
      V[2][2]);

  const uint32_t COMP_D_ITERS = 10;
  glm::vec3 D(0.0f);
  for (uint32_t i = 0; i < COMP_D_ITERS; ++i) {
    glm::vec3 sigmaPlusD = sigma + D;
    float product = sigmaPlusD.x * sigmaPlusD.y * sigmaPlusD.z;
    float omega = glm::clamp(product, tet.minOmega, tet.maxOmega);
    float C = product - omega;
    glm::vec3 gradC(
        sigmaPlusD.y * sigmaPlusD.z,
        sigmaPlusD.x * sigmaPlusD.z,
        sigmaPlusD.x * sigmaPlusD.y);
    D = (glm::dot(gradC, D) - C) * gradC / glm::dot(gradC, gradC);
  }

  sigma += D;

  for (uint32_t i = 0; i < 3; ++i) {
    sigma[i] = glm::clamp(sigma[i], tet.minStrain, tet.maxStrain);
  }

  if (glm::determinant(F) < 0.0f) {
    sigma[2] *= -1.0f;
  }

  // The "fixed" deformation gradient
  glm::mat3
      sigma_(sigma.x, 0.0f, 0.0f, 0.0f, sigma.y, 0.0f, 0.0f, 0.0f, sigma.z);
  glm::mat3 Fhat = U * sigma_ * glm::transpose(V);

  glm::mat3x4 P1(0.0f);
  P1[0][1] = Fhat[0][0];
  P1[1][1] = Fhat[1][0];
  P1[2][1] = Fhat[2][0];

  P1[0][2] = Fhat[0][1];
  P1[1][2] = Fhat[1][1];
  P1[2][2] = Fhat[2][1];

  P1[0][3] = Fhat[0][2];
  P1[1][3] = Fhat[1][2];
  P1[2][3] = Fhat[2][2];

  wAtBp[i] = tet.w * tet.AtB * P1;
}

namespace Pies {
TetrahedralConstraint::TetrahedralConstraint(
    const Node& a,
    const Node& b,
    const Node& c,
    const Node& d,
    float w_,
    float minStrain_,
    float maxStrain_,
    float minOmega_,
    float maxOmega_)
    : nodeIds{static_cast<int>(a.id), static_cast<int>(b.id), static_cast<int>(c.id), static_cast<int>(d.id)},
      w(w_),
      minStrain(minStrain_),
      maxStrain(maxStrain_),
      minOmega(minOmega_),
      maxOmega(maxOmega_) {

  // Converts world positions to differential coords
  glm::mat4x3 worldToDiff(0.0f);
  worldToDiff[0][0] = -1.0f;
  worldToDiff[0][1] = -1.0f;
  worldToDiff[0][2] = -1.0f;

  worldToDiff[1][0] = 1.0f;
  worldToDiff[2][1] = 1.0f;
  worldToDiff[3][2] = 1.0f;

  // Converts barycentric coords to world differential cords
  glm::mat3 baryToDiff(
      b.position - a.position,
      c.position - a.position,
      d.position - a.position);
  this->Qinv = glm::inverse(baryToDiff);

  // TODO: Simplify??
  glm::mat3x4 At = glm::transpose(glm::transpose(this->Qinv) * worldToDiff);
  // Add empty row corresponding to the first row since its differential
  // coord is always 0 (can we just ignore this?)
  // Note: Be careful about the column/row indices if changing this in the
  // future.
  this->AtB = glm::mat4(0.0f);
  this->AtB[1] = At[0];
  this->AtB[2] = At[1];
  this->AtB[3] = At[2];
}

TetrahedralConstraintCollection::TetrahedralConstraintCollection(
    std::vector<TetrahedralConstraint>&& tetConstraints,
    Eigen::SparseMatrix<float>& systemMatrix)
    : _tets(std::move(tetConstraints)) {

  this->_wAtBp.resize(this->_tets.size());

  for (const TetrahedralConstraint& tet : this->_tets) {
    // B is identity here
    glm::mat4 AtA = tet.AtB * glm::transpose(tet.AtB);
    for (int i = 0; i < 4; ++i) {
      int nodeId_i = tet.nodeIds[i];
      for (int j = 0; j < 4; ++j) {
        int nodeId_j = tet.nodeIds[j];
        systemMatrix.coeffRef(nodeId_i, nodeId_j) += tet.w * AtA[j][i];
      }
    }
  }

  // Create device memory for the tet constraints
  hipMalloc(
      &this->_dev_tets,
      sizeof(TetrahedralConstraint) * this->_tets.size());
  hipMemcpy(
      this->_dev_tets,
      this->_tets.data(),
      sizeof(TetrahedralConstraint) * this->_tets.size(),
      hipMemcpyHostToDevice);

  // Create device memory to hold the projection output.
  hipMalloc(&this->_dev_wAtBp, sizeof(glm::mat3x4) * this->_tets.size());
}

TetrahedralConstraintCollection::TetrahedralConstraintCollection(
    TetrahedralConstraintCollection&& rhs)
    : _tets(std::move(rhs._tets)),
      _dev_tets(rhs._dev_tets),
      _dev_wAtBp(rhs._dev_wAtBp),
      _wAtBp(std::move(rhs._wAtBp)) {
  rhs._dev_tets = nullptr;
  rhs._dev_wAtBp = nullptr;
}

TetrahedralConstraintCollection& TetrahedralConstraintCollection::operator=(
    TetrahedralConstraintCollection&& rhs) {
  this->_tets = std::move(rhs._tets);
  this->_dev_tets = rhs._dev_tets;
  this->_dev_wAtBp = rhs._dev_wAtBp;
  this->_wAtBp = std::move(rhs._wAtBp);

  rhs._dev_tets = nullptr;
  rhs._dev_wAtBp = nullptr;

  return *this;
}

TetrahedralConstraintCollection::~TetrahedralConstraintCollection() {
  if (this->_dev_tets) {
    hipFree(this->_dev_tets);
  }

  if (this->_dev_wAtBp) {
    hipFree(this->_dev_wAtBp);
  }
}

void TetrahedralConstraintCollection::project(glm::vec3* devNodePositions) {
  int tetCount = static_cast<int>(this->_tets.size());

  if (tetCount == 0) {
    return;
  }

  int threadCount = 256;
  int pblks = int((tetCount + threadCount - 1) / threadCount);
  projectTets<<<pblks, threadCount>>>(
      this->_dev_tets,
      devNodePositions,
      this->_dev_wAtBp,
      tetCount);

  hipMemcpy(
      this->_wAtBp.data(),
      this->_dev_wAtBp,
      sizeof(glm::mat3x4) * this->_tets.size(),
      hipMemcpyDeviceToHost);
}

void TetrahedralConstraintCollection::setupGlobalForceVector(
    Eigen::MatrixXf& forceVector) const {
  for (size_t tetId = 0; tetId < this->_tets.size(); ++tetId) {
    const TetrahedralConstraint& tet = this->_tets[tetId];
    const glm::mat3x4& wAtBp = this->_wAtBp[tetId];
    for (uint32_t i = 0; i < 4; ++i) {
      uint32_t nodeId_i = tet.nodeIds[i];
      forceVector.coeffRef(nodeId_i, 0) += wAtBp[0][i];
      forceVector.coeffRef(nodeId_i, 1) += wAtBp[1][i];
      forceVector.coeffRef(nodeId_i, 2) += wAtBp[2][i];
    }
  }
}
} // namespace Pies