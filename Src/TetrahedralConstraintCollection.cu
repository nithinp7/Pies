#include "hip/hip_runtime.h"

#include "TetrahedralConstraintCollection.h"

#include <svd3x3/svd3_cuda.h>

#include "hip/hip_runtime.h"
#include ""

#define GLM_FORCE_CUDA
#include <glm/glm.hpp>
#include <glm/gtc/matrix_access.hpp>
#include <glm/gtc/matrix_inverse.hpp>
#include <glm/gtc/matrix_transform.hpp>

__global__ void projectTets(
    Pies::TetrahedralConstraint* devTets,
    glm::vec3* devNodePositions,
    Eigen::Matrix<float, 4, 3>* wAtBp,
    int tetCount) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= tetCount) {
    return;
  }

  // TODO: Avoid the awkward switching back and forth between
  // Eigen and glm here

  const Pies::TetrahedralConstraint& tet = devTets[i];

  const glm::vec3& x1 = devNodePositions[tet.nodeIds[0]];
  const glm::vec3& x2 = devNodePositions[tet.nodeIds[1]];
  const glm::vec3& x3 = devNodePositions[tet.nodeIds[2]];
  const glm::vec3& x4 = devNodePositions[tet.nodeIds[3]];

  glm::mat3 P(x2 - x1, x3 - x1, x4 - x1);

  // Deformation gradient
  glm::mat3 F = P * tet.Qinv;

  Eigen::Matrix3f F_;
  F_ << F[0][0], F[0][1], F[0][2], F[1][0], F[1][1], F[1][2], F[2][0], F[2][1],
      F[2][2];

  Eigen::Matrix3f U;
  Eigen::Vector3f singularValues;
  Eigen::Matrix3f V;

  svd(
      // Remember glm is stored column major
      F[0][0],
      F[1][0],
      F[2][0],
      F[0][1],
      F[1][1],
      F[2][1],
      F[0][2],
      F[1][2],
      F[2][2],
      U.coeffRef(0, 0),
      U.coeffRef(0, 1),
      U.coeffRef(0, 2),
      U.coeffRef(1, 0),
      U.coeffRef(1, 1),
      U.coeffRef(1, 2),
      U.coeffRef(2, 0),
      U.coeffRef(2, 1),
      U.coeffRef(2, 2),
      singularValues[0],
      singularValues[1],
      singularValues[2],
      V.coeffRef(0, 0),
      V.coeffRef(0, 1),
      V.coeffRef(0, 2),
      V.coeffRef(1, 0),
      V.coeffRef(1, 1),
      V.coeffRef(1, 2),
      V.coeffRef(2, 0),
      V.coeffRef(2, 1),
      V.coeffRef(2, 2));

  glm::vec3 sigma(singularValues[0], singularValues[1], singularValues[2]);

  const uint32_t COMP_D_ITERS = 10;
  glm::vec3 D(0.0f);
  for (uint32_t i = 0; i < COMP_D_ITERS; ++i) {
    glm::vec3 sigmaPlusD = sigma + D;
    float product = sigmaPlusD.x * sigmaPlusD.y * sigmaPlusD.z;
    float omega = glm::clamp(product, tet.minOmega, tet.maxOmega);
    float C = product - omega;
    glm::vec3 gradC(
        sigmaPlusD.y * sigmaPlusD.z,
        sigmaPlusD.x * sigmaPlusD.z,
        sigmaPlusD.x * sigmaPlusD.y);
    D = (glm::dot(gradC, D) - C) * gradC / glm::dot(gradC, gradC);
  }

  singularValues[0] += D.x;
  singularValues[1] += D.y;
  singularValues[2] += D.z;

  for (uint32_t i = 0; i < 3; ++i) {
    singularValues[i] =
        glm::clamp(singularValues[i], tet.minStrain, tet.maxStrain);
  }

  if (glm::determinant(F) < 0.0f) {
    singularValues[2] *= -1.0f;
  }

  // The "fixed" deformation gradient
  Eigen::Matrix3f Fhat =
      U * singularValues.asDiagonal() * V.transpose();

  Eigen::Matrix<float, 4, 3> P1 = Eigen::Matrix<float, 4, 3>::Zero();
  // TODO: Fhat.transpose()???
  P1.block(1, 0, 3, 3) = Fhat;

  wAtBp[i] = tet.w * tet.AtB * P1;
}

namespace Pies {
TetrahedralConstraint::TetrahedralConstraint(
    const Node& a,
    const Node& b,
    const Node& c,
    const Node& d,
    float w_,
    float minStrain_,
    float maxStrain_,
    float minOmega_,
    float maxOmega_)
    : nodeIds{static_cast<int>(a.id), static_cast<int>(b.id), static_cast<int>(c.id), static_cast<int>(d.id)},
      w(w_),
      minStrain(minStrain_),
      maxStrain(maxStrain_),
      minOmega(minOmega_),
      maxOmega(maxOmega_) {

  // Converts world positions to differential coords
  Eigen::Matrix<float, 3, 4> worldToDiff = Eigen::Matrix<float, 3, 4>::Zero();
  worldToDiff.coeffRef(0, 0) = -1.0f;
  worldToDiff.coeffRef(1, 0) = -1.0f;
  worldToDiff.coeffRef(2, 0) = -1.0f;

  worldToDiff.coeffRef(0, 1) = 1.0f;
  worldToDiff.coeffRef(1, 2) = 1.0f;
  worldToDiff.coeffRef(2, 3) = 1.0f;

  // Converts barycentric coords to world differential cords
  glm::mat3 baryToDiff(
      b.position - a.position,
      c.position - a.position,
      d.position - a.position);
  glm::mat3 diffToBary = glm::inverse(baryToDiff);

  Eigen::Matrix3f diffToBary_;
  diffToBary_ << diffToBary[0][0], diffToBary[0][1], diffToBary[0][2],
      diffToBary[1][0], diffToBary[1][1], diffToBary[1][2], diffToBary[2][0],
      diffToBary[2][1], diffToBary[2][2];

  Eigen::Matrix<float, 3, 4> A_ = diffToBary_ * worldToDiff;
  Eigen::Matrix4f A = Eigen::Matrix4f::Zero();
  // A.coeffRef(1, 0) = -1.0f;
  // A.coeffRef(2, 0) = -1.0f;
  // A.coeffRef(3, 0) = -1.0f;

  // A.coeffRef(1, 1) = 1.0f;
  // A.coeffRef(2, 2) = 1.0f;
  // A.coeffRef(3, 3) = 1.0f;

  A.row(0) << 0.0f, 0.0f, 0.0f, 0.0f;
  A.row(1) = A_.row(0);
  A.row(2) = A_.row(1);
  A.row(3) = A_.row(2);

  // B is identity

  this->Qinv = diffToBary;
  this->AtB = A;
}

TetrahedralConstraintCollection::TetrahedralConstraintCollection(
    std::vector<TetrahedralConstraint>&& tetConstraints,
    Eigen::SparseMatrix<float>& systemMatrix)
    : _tets(std::move(tetConstraints)) {

  this->_wAtBp.resize(this->_tets.size());

  for (const TetrahedralConstraint& tet : this->_tets) {
    // B is identity here
    Eigen::Matrix<float, 4, 4> AtA = tet.AtB.transpose() * tet.AtB;
    for (int i = 0; i < 4; ++i) {
      int nodeId_i = tet.nodeIds[i];
      for (int j = 0; j < 4; ++j) {
        int nodeId_j = tet.nodeIds[j];
        systemMatrix.coeffRef(nodeId_i, nodeId_j) += tet.w * AtA.coeff(i, j);
      }
    }
  }

  // Create device memory for the tet constraints
  hipMalloc(
      &this->_dev_tets,
      sizeof(TetrahedralConstraint) * this->_tets.size());
  hipMemcpy(
      this->_dev_tets,
      this->_tets.data(),
      sizeof(TetrahedralConstraint) * this->_tets.size(),
      hipMemcpyHostToDevice);

  // Create device memory to hold the projection output.
  hipMalloc(
      &this->_dev_wAtBp,
      sizeof(Eigen::Matrix<float, 4, 3>) * this->_tets.size());
}

TetrahedralConstraintCollection::~TetrahedralConstraintCollection() {
  hipFree(this->_dev_tets);
  hipFree(this->_dev_wAtBp);
}

void TetrahedralConstraintCollection::project(glm::vec3* devNodePositions) {
  int tetCount = static_cast<int>(this->_tets.size());

  int threadCount = 504;
  int pblks = int((tetCount + threadCount - 1) / threadCount);
  projectTets<<<pblks, threadCount>>>(
      this->_dev_tets,
      devNodePositions,
      this->_dev_wAtBp,
      tetCount);

  hipMemcpy(
      this->_wAtBp.data(),
      this->_dev_wAtBp,
      sizeof(Eigen::Matrix<float, 4, 3>) * this->_tets.size(),
      hipMemcpyDeviceToHost);
}

void TetrahedralConstraintCollection::setupGlobalForceVector(
    Eigen::MatrixXf& forceVector) const {
  for (size_t tetId = 0; tetId < this->_tets.size(); ++tetId) {
    const TetrahedralConstraint& tet = this->_tets[tetId];
    const Eigen::Matrix<float, 4, 3>& wAtBp = this->_wAtBp[tetId];
    for (uint32_t i = 0; i < 4; ++i) {
      uint32_t nodeId_i = tet.nodeIds[i];
      forceVector.coeffRef(nodeId_i, 0) += wAtBp.coeff(i, 0);
      forceVector.coeffRef(nodeId_i, 1) += wAtBp.coeff(i, 1);
      forceVector.coeffRef(nodeId_i, 2) += wAtBp.coeff(i, 2);
    }
  }
}
} // namespace Pies