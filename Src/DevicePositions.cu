#include "DevicePositions.h"
#include "Node.h"

#include "hip/hip_runtime.h"
#include ""

#include <cassert>

namespace Pies {
DevicePositions::DevicePositions(size_t count) 
  : _count(count) {
  this->_scratch.resize(count);
  hipError_t err = hipMalloc(&this->_devPositions, sizeof(glm::vec3) * count);
  
  hipDeviceSynchronize();
}

DevicePositions::~DevicePositions() {
  if (this->_devPositions) {
    hipFree(this->_devPositions);
  }

  hipDeviceSynchronize();
}

DevicePositions::DevicePositions(DevicePositions&& rhs) 
  : _count(rhs._count),
    _devPositions(rhs._devPositions),
    _scratch(std::move(rhs._scratch)) {
  rhs._devPositions = nullptr;
  rhs._count = 0;
}

DevicePositions& DevicePositions::operator=(DevicePositions&& rhs) {
  this->_count = rhs._count;
  this->_devPositions = rhs._devPositions;
  this->_scratch = std::move(rhs._scratch);

  rhs._devPositions = nullptr;
  rhs._count = 0;

  return *this;
}

void DevicePositions::upload(const std::vector<Node>& nodes) {
  assert(nodes.size() == this->_count);

  if (nodes.size() == 0) {
    return;
  }
  
  // Scalarize / parallelize this??
  for (size_t i = 0; i < this->_count; ++i) {
    this->_scratch[i] = nodes[i].position;
  }

  hipMemcpy(this->_devPositions, this->_scratch.data(), sizeof(glm::vec3) * this->_count, hipMemcpyHostToDevice);
  
  hipDeviceSynchronize();
}
} // namespace Pies